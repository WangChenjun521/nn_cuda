
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add2_kernel(float* c,
                            const float* a,
                            const float* b,
                            int n) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
            i < n; i += gridDim.x * blockDim.x) {
        c[i] = a[i] + b[i];
    }
}
__global__ void helloFromGPU(void)
{
  printf("Hello World from GPU！\n");
}

__global__ void VecAdd(int* A, int* B, int* C)
{
    for(int j=0 ;j<100000000;j++){
        int i = threadIdx.x;
        C[i] = A[i] + B[i];
    }
    
}

void launch_add2(float* c,
                 const float* a,
                 const float* b,
                 int n) {
    dim3 grid((n + 1023) / 1024);
    dim3 block(1024);
    // helloFromGPU<<<1,10>>>();

    add2_kernel<<<grid, block>>>(c, a, b, n);
    // const int N=5;
    // int A[N]={1,2,3,4,5};
    // int B[N]={2,2,2,2,2};
    // int C[N]={0};

    // int *dev_a = 0;
    // int *dev_b = 0;
    // int *dev_c = 0;

    // cudaSetDevice(0);
    // cudaMalloc((void**)&dev_c, N * sizeof(int));
    // cudaMalloc((void**)&dev_a, N * sizeof(int));
    // cudaMalloc((void**)&dev_b, N * sizeof(int));
    // cudaMemcpy(dev_a, A, N * sizeof(int), cudaMemcpyHostToDevice);
    // cudaMemcpy(dev_b, B, N * sizeof(int), cudaMemcpyHostToDevice);

    // VecAdd<<<1, N>>>(dev_a, dev_b, dev_c);

    // cudaGetLastError();
    // cudaDeviceSynchronize();
    // cudaMemcpy(C, dev_c, N * sizeof(int), cudaMemcpyDeviceToHost);
    // cudaFree(dev_c);
    // cudaFree(dev_a);
    // cudaFree(dev_b);

    // for (int i = 0; i < N; i++)
    // {
    //     if (i!=0) printf(" ");
    //     printf("%d",C[i]);
    //     if (i==N-1)printf("\n");
    // }
    
}